#include "hip/hip_runtime.h"
// kernel.cu
//This is the project for optimizing memory access stride.
//-------------------------------------------------------------------
#include "header.h"
#define SHARED_TEST
#include <time.h>
#include <stdlib.h>

#define N 16
#define BLK_SIZE 4 //blocksize(ThreadDim)
#define WIN_SIZE 2

#ifdef OLD_CODE
__global__ void compute_win2D(int *knode, int *kcell)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;

	__shared__ int smem[BLK_SIZE + WIN_SIZE][BLK_SIZE + WIN_SIZE];
	//printf("%d %d %d %d\n", threadIdx.y, threadIdx.x, smem[threadIdx.y][threadIdx.x], *((int*)kcell + (idx)* N + idy));
	int temp, x, y;
	//if ((threadIdx.x < N) && (threadIdx.y < N))
	{
			//find minimum in window
			smem[threadIdx.y][threadIdx.x] = *((int*)kcell + idx * N + idy);
			if (threadIdx.x >= (N - WIN_SIZE))
			{
				//smem[threadIdx.x][threadIdx.y + WIN_SIZE] = *((int*)kcell + idx * N + idy + WIN_SIZE);
				smem[threadIdx.y][threadIdx.x + WIN_SIZE] = *((int*)kcell + idx * N + idy);
				printf("%d %d %d %d\n", threadIdx.y, threadIdx.x + WIN_SIZE, smem[threadIdx.y][threadIdx.x + WIN_SIZE], *((int*)kcell + (idx)* N + idy));
			}

			if (threadIdx.y >= (N - WIN_SIZE))
			{
				//smem[threadIdx.x + WIN_SIZE][threadIdx.y] = *((int*)kcell + (idx + WIN_SIZE) * N + idy);
				smem[threadIdx.y + WIN_SIZE][threadIdx.x] = *((int*)kcell + (idx)* N + idy);
				printf("%d %d %d %d\n", threadIdx.y + WIN_SIZE, threadIdx.x, smem[threadIdx.y + WIN_SIZE][threadIdx.x], *((int*)kcell + (idx)* N + idy));
			}

			if ((threadIdx.x >= (N - WIN_SIZE)))
			{
					if ((threadIdx.y >= (N - WIN_SIZE)))
					{
						//smem[threadIdx.x + WIN_SIZE][threadIdx.y + WIN_SIZE] = *((int*)kcell + (idx + WIN_SIZE) * N + idy + WIN_SIZE);
						smem[threadIdx.y + WIN_SIZE][threadIdx.x + WIN_SIZE] = *((int*)kcell + (idx)* N + idy);
						//*((int*)smem + (threadIdx.x + WIN_SIZE)*N + threadIdx.y + WIN_SIZE) = *((int*)kcell + (idx)* N + idy);
						printf("%d %d %d %d\n", threadIdx.y + WIN_SIZE, threadIdx.x + WIN_SIZE, smem[threadIdx.y + WIN_SIZE][threadIdx.x + WIN_SIZE], *((int*)kcell + (idx)* N + idy));

					}
			}
			__syncthreads();

			//temp = *((int *)kcell + idx*N + idy);
			temp = smem[threadIdx.y][threadIdx.x];
			//temp = 0;
			//printf("%d %d %d\n", idx, idy, temp);
			//printf("%x ", (int *)kcell + idx*N + idy);
			for (y = 0; y < WIN_SIZE; y++)
			{ 
				for (x = 0; x < WIN_SIZE; x++)
				{
					//if (temp > *((int *)smem + (threadIdx.x + x) *N + (threadIdx.y + y)))
					//{
					//	temp = *((int *)smem + (threadIdx.x + x) *N + (threadIdx.y + y));
					//}
					if (temp > smem[threadIdx.y+y][threadIdx.x+x])
					{
						temp = smem[threadIdx.y + y][threadIdx.x + x];
					}
				}
		  }
			*((int *)knode + idx*N + idy) = temp;
	}
	printf("%d %d %d\n", idx, idy, temp);
}
//-------------------------------------------------------------------
//__global__ void compute_win2D(int *knode, int *kcell)
//{
//	int idx = blockIdx.x*blockDim.x + threadIdx.x;
//	int idy = blockIdx.y*blockDim.y + threadIdx.y;
//	int temp, x, y;
//	*((int *)knode + idx*N + idy) = *((int *)kcell + idx*N + idy);
//	__syncthreads();
//	if ((threadIdx.x < N) && (threadIdx.y < N))
//	{
//		if (threadIdx.x >= N - WIN_SIZE)
//		{
//			*((int *)knode + (threadIdx.x + WIN_SIZE)*N + threadIdx.y) = *((int *)kcell + (idx)*N + idy);
//			printf("%d %d %d\n", idx + WIN_SIZE, idy, *((int *)knode + idx*N + idy));
//		}
//		if (idy >= N - WIN_SIZE)
//		{
//			*((int *)knode + (threadIdx.x)*N + threadIdx.y + WIN_SIZE) = *((int *)kcell + (idx)*N + idy);
//			printf("%d %d %d\n", idx, idy + WIN_SIZE, *((int *)knode + idx*N + idy));
//		}
//
//		if (threadIdx.x >= N - WIN_SIZE)
//		{
//			if (idy >= N - WIN_SIZE)
//			{
//				*((int *)knode + (threadIdx.x + WIN_SIZE)*N + threadIdx.y + WIN_SIZE) = *((int *)kcell + (idx)*N + idy);
//				printf("%d %d %d\n", idx + WIN_SIZE, idy + WIN_SIZE, *((int *)knode + idx*N + idy));
//			}
//		}
//		__syncthreads();
//		printf("%d %d %d\n", idx, idy, *((int *)knode + idx*N + idy));
//		printf("/");
//
//		//find minimum in window
//		temp = *((int *)knode + idx*N + idy);
//		for (x = 0; x < WIN_SIZE; x++)
//		{
//			for (y = 0; y < WIN_SIZE; y++)
//			{
//					if (temp  > *((int *)knode + (idx + x) *N + (idy + y)))
//					{
//						temp = *((int *)knode + (idx + x) *N + (idy + y));
//					}
//			}
//		}
//		//__syncthreads();
//		*((int *)knode + idx*N + idy) = temp;
//	}
//	printf("\n%d %d %d\n", idx, idy, temp);
//}
#endif
//ORIGINAL KERNEL FOR REFERENCE
#if 0
__global__ void A5_fast_lo_stats_kernel(float* xVal, float* outStd, float* outSkw, float* outKrt)
{
	//Declarations
	float xVal_local[256];
	float mean = 0, stdev = 0, skw = 0, krt = 0, stmp = 0;
	int iB, jB;
	//https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
	//int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	//int threadId = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x) + threadIdx.x;
	int i = 4 * (threadIdx.x + blockIdx.x * blockDim.x);
	int j = 4 * (threadIdx.y + blockIdx.y * blockDim.y);
	if ((i < 497) && (j < 497))//512-15=497
	{
		//for (j = 0; j<512 - 15; j += 4)
		// THE FOLLOWING SET OF RUNNING SUMS CAN BE A set of PARALLEL REDUCTIONs (in shared memory?)
		// 256 itteratios -> log2(256)=8 itterations
		// Store block into registers (256 x 4Bytes = 1kB)
		int idx = 0;
		for (iB = i; iB < i + 16; iB++)
		{
			for (jB = j; jB < j + 16; jB++)
			{
				//xVal_local[idx] = xVal[iB * 512 + jB];
				xVal_local[idx] = xVal[iB * 512 + jB];
				//printf("%d ", iB * 512 + jB);
				idx++;
			}
		}
		//Traverse through and get mean
		float mean = 0;
		for (idx = 0; idx < 256; idx++)
			mean += xVal_local[idx];				//this can be a simple reduction in shared memory
		mean = mean / 256.0f;

		//Traverse through and get stdev, skew and kurtosis
		stdev = 0;
		skw = 0;
		krt = 0;
		float xV_mean = 0;
		for (idx = 0; idx < 256; idx++)
		{
			// Place this commonly re-used value into a register to preserve temporal localitiy
			xV_mean = xVal_local[idx] - mean;
			stdev = stdev + (xV_mean * xV_mean);
			skw = skw + (xV_mean * xV_mean * xV_mean);
			krt = krt + (xV_mean * xV_mean * xV_mean * xV_mean);
		}
		stmp = sqrt(stdev / 256.0f);
		stdev = sqrt(stdev / 255.0f);//MATLAB's std is a bit different
		/*
		if (i + j <5)
		{
		printf("%f %f %f %f %f \n", stdev,stmp,stdev, skw, krt);
		}
		*/
		if (stmp != 0){
			skw = (skw / 256.0f) / ((stmp)*(stmp)*(stmp));
			krt = (krt / 256.0f) / ((stmp)*(stmp)*(stmp)*(stmp));
		}
		else{
			skw = 0;
			krt = 0;
		}
		/*
		if (i + j <5)
		{
		printf("%f %f \n", skw, krt);
		}*/
		//---------------------------------------------------------------------------
		// This is the nearest neighbor interpolation - ACTUALLY NOT NEEDED!!!!!!!!
		// To remove the nested for loop here we need to modifie the algorithm to 
		// adjust for the pointwise muliplication done far later that uses a
		// 512x512 dimension matrix derived from the matrices this kernel produces
		// The modified output would be PxP (as described mathematically in the paper).
		//---------------------------------------------------------------------------
		// Only this final output should be written to global memory:

		for (iB = i; iB < i + 4; iB++)
		{
			for (jB = j; jB < j + 4; jB++)
			{
				// Added if-else statement here:
				if (i > 500 || j > 500)
				{
					outStd[(iB * 512) + jB] = 0;
					outSkw[(iB * 512) + jB] = 0;
					outKrt[(iB * 512) + jB] = 0;
				}
				else
				{
					outStd[(iB * 512) + jB] = stdev;
					outSkw[(iB * 512) + jB] = skw;
					outKrt[(iB * 512) + jB] = krt;
				}
				// Added if-else statement here:
				//if (i > 500 || j > 500)
				//{
				//	outStd[threadId] = 0;
				//	outSkw[threadId] = 0;
				//	outKrt[threadId] = 0;
				//}
				//else
				//{
				//	outStd[threadId] = stdev;
				//	outSkw[threadId] = skw;
				//	outKrt[threadId] = krt;
				//}
			}
		}
	}
}
#endif
// Kernel Wrapper
__global__ void A5_fast_lo_stats_kernel(float* xVal, float* out)
{
	//Declarations
	float xVal_local[256];
	float mean = 0;
	int iB, jB;
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x) + threadIdx.x;
	int i = 4 * (threadIdx.x + blockIdx.x * blockDim.x);
	int j = 4 * (threadIdx.y + blockIdx.y * blockDim.y);
	if ((i < 497) && (j < 497))//512-15=497
	{
		//for (j = 0; j<512 - 15; j += 4)
		// THE FOLLOWING SET OF RUNNING SUMS CAN BE A set of PARALLEL REDUCTIONs (in shared memory?)
		// 256 itteratios -> log2(256)=8 itterations
		// Store block into registers (256 x 4Bytes = 1kB)
		int idx = 0;
		for (iB = i; iB < i + 16; iB++)
		{
			for (jB = j; jB < j + 16; jB++)
			{
				//xVal_local[idx] = xVal[iB * 512 + jB];
				xVal_local[idx] = xVal[iB * 512 + jB];
				//printf("%d ", iB * 512 + jB);
				idx++;
			}
		}
		//Traverse through and get mean
		float mean = 0;
		for (idx = 0; idx < 256; idx++)
			mean += xVal_local[idx];				//this can be a simple reduction in shared memory
		mean = mean / 256.0f;
		out[threadId] = mean;
	}
}
__global__ void A5_fast_lo_stats_kernel_SHARED(float* xVal, float* xVal_window,  float* out)
{
	//Declarations
	//__shared__ float xVal_Shm[256];
	__shared__ float xVal_smem[BLK_SIZE + WIN_SIZE][BLK_SIZE + WIN_SIZE]; //Size = (16 + 16) * (16 + 16) * 4 bytes
	float mean = 0, stdev = 0, skw = 0, krt = 0, stmp = 0;
	int iB, jB;
	//https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x) + threadIdx.x;
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	
	//Used in code
	int global_idx = 4 * (threadIdx.x + blockIdx.x * blockDim.x);
	int global_idy = 4 * (threadIdx.y + blockIdx.y * blockDim.y);

	if ((global_idx < N - 15) && (global_idy < N - 15))//512-15=497
	{
		//for (j = 0; j<512 - 15; j += 4)
		// THE FOLLOWING SET OF RUNNING SUMS CAN BE A set of PARALLEL REDUCTIONs (in shared memory?)
		// 256 itteratios -> log2(256)=8 itterations
			//find minimum in window
			xVal_smem[threadIdx.x][threadIdx.y] = xVal[idx * N + idy];
			if (threadIdx.y > (N - WIN_SIZE))
			{
				//smem[threadIdx.x][threadIdx.y + WIN_SIZE] = *((int*)kcell + idx * N + idy + WIN_SIZE);
				xVal_smem[threadIdx.x][threadIdx.y + WIN_SIZE] = xVal[idx * N + idy + WIN_SIZE];
				//printf("%d %d %d %d\n", threadIdx.y, threadIdx.x + WIN_SIZE, xVal_smem[threadIdx.y][threadIdx.x + WIN_SIZE], *((int*)kcell + (idx)* N + idy));
			}

			if (threadIdx.x > (N - WIN_SIZE))
			{
				//smem[threadIdx.x + WIN_SIZE][threadIdx.y] = *((int*)kcell + (idx + WIN_SIZE) * N + idy);
				xVal_smem[threadIdx.x + WIN_SIZE][threadIdx.y] = xVal[(idx +WIN_SIZE)* N + idy];
				//printf("%d %d %d %d\n", threadIdx.y + WIN_SIZE, threadIdx.x, xVal_smem[threadIdx.y + WIN_SIZE][threadIdx.x], *((int*)kcell + (idx)* N + idy));
			}

			if ((threadIdx.x > (N - WIN_SIZE)))
			{
				if ((threadIdx.y > (N - WIN_SIZE)))
				{
					//smem[threadIdx.x + WIN_SIZE][threadIdx.y + WIN_SIZE] = *((int*)kcell + (idx + WIN_SIZE) * N + idy + WIN_SIZE);
					xVal_smem[threadIdx.x + WIN_SIZE][threadIdx.y + WIN_SIZE] = xVal[(idx + WIN_SIZE)* N + idy + WIN_SIZE];
					//*((int*)smem + (threadIdx.x + WIN_SIZE)*N + threadIdx.y + WIN_SIZE) = *((int*)kcell + (idx)* N + idy);
					//printf("%d %d %d %d\n", threadIdx.y + WIN_SIZE, threadIdx.x + WIN_SIZE, xVal_smem[threadIdx.y + WIN_SIZE][threadIdx.x + WIN_SIZE], *((int*)kcell + (idx)* N + idy));

				} 
			}
			__syncthreads();

			//temp = *((int *)kcell + idx*N + idy);
			mean = 0;
			//printf("%d %d %d\n", idx, idy, temp);
			//printf("%x ", (int *)kcell + idx*N + idy);
			for (int y = 0; y < WIN_SIZE; y++)
			{
				for (int x = 0; x < WIN_SIZE; x++)
				{
					mean += xVal_smem[threadIdx.y + y][threadIdx.x + x];
				}
			}
			mean = mean / 256.0f;
			out[threadId] = mean;
		}
	}
#ifdef OLD_CODE
__global__ void A5_fast_lo_stats_kernel_SHARED(float* xVal, float* out)
{
	//Declarations
	//__shared__ float xVal_Shm[256];
	__shared__ float xVal_smem[BLK_SIZE + WIN_SIZE][BLK_SIZE + WIN_SIZE]; //Size = (16 + 16) * (16 + 16) * 4 bytes
	float mean = 0, stdev = 0, skw = 0, krt = 0, stmp = 0;
	int iB, jB;
	//https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int global_idx = 4 * (threadIdx.x + blockIdx.x * blockDim.x);
	int global_idy = 4 * (threadIdx.y + blockIdx.y * blockDim.y);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	if ((global_idx < N - 15) && (global_idy < N - 15))//512-15=497
	{
		//for (j = 0; j<512 - 15; j += 4)
		// THE FOLLOWING SET OF RUNNING SUMS CAN BE A set of PARALLEL REDUCTIONs (in shared memory?)
		// 256 itteratios -> log2(256)=8 itterations
		if ((threadIdx.x < BLK_SIZE) && (threadIdx.y < BLK_SIZE)) //Re-check the need for this if statement
		{
			//find minimum in window
			xVal_smem[threadIdx.y][threadIdx.x] = xVal[idx * N + idy];
			if (threadIdx.x >= (BLK_SIZE - WIN_SIZE))
			{
				//smem[threadIdx.x][threadIdx.y + WIN_SIZE] = *((int*)kcell + idx * N + idy + WIN_SIZE);
				xVal_smem[threadIdx.y][threadIdx.x + WIN_SIZE] = xVal[idx * N + idy];
				//printf("%d %d %d %d\n", threadIdx.y, threadIdx.x + WIN_SIZE, xVal_smem[threadIdx.y][threadIdx.x + WIN_SIZE], *((int*)kcell + (idx)* N + idy));
			}

			if (threadIdx.y >= (N - WIN_SIZE))
			{
				//smem[threadIdx.x + WIN_SIZE][threadIdx.y] = *((int*)kcell + (idx + WIN_SIZE) * N + idy);
				xVal_smem[threadIdx.y + WIN_SIZE][threadIdx.x] = xVal[idx * N + idy];
				//printf("%d %d %d %d\n", threadIdx.y + WIN_SIZE, threadIdx.x, xVal_smem[threadIdx.y + WIN_SIZE][threadIdx.x], *((int*)kcell + (idx)* N + idy));
			}

			if ((threadIdx.x >= (N - WIN_SIZE)))
			{
				if ((threadIdx.y >= (N - WIN_SIZE)))
				{
					//smem[threadIdx.x + WIN_SIZE][threadIdx.y + WIN_SIZE] = *((int*)kcell + (idx + WIN_SIZE) * N + idy + WIN_SIZE);
					xVal_smem[threadIdx.y + WIN_SIZE][threadIdx.x + WIN_SIZE] = xVal[idx * N + idy];
					//*((int*)smem + (threadIdx.x + WIN_SIZE)*N + threadIdx.y + WIN_SIZE) = *((int*)kcell + (idx)* N + idy);
					//printf("%d %d %d %d\n", threadIdx.y + WIN_SIZE, threadIdx.x + WIN_SIZE, xVal_smem[threadIdx.y + WIN_SIZE][threadIdx.x + WIN_SIZE], *((int*)kcell + (idx)* N + idy));

				}
			}
			__syncthreads();

			//temp = *((int *)kcell + idx*N + idy);
			mean = 0;
			//printf("%d %d %d\n", idx, idy, temp);
			//printf("%x ", (int *)kcell + idx*N + idy);
			for (int y = 0; y < WIN_SIZE; y++)
			{
				for (int x = 0; x < WIN_SIZE; x++)
				{
					mean += xVal_smem[threadIdx.y + y][threadIdx.x + x];
				}
			}
			mean = mean / 256.0f;
			out[threadId] = mean;
		}
	}
}
#endif
void write_to_file_DEBUG(float* w, int size)
{
	std::ofstream outFile;
	outFile.open("TEST.txt");
	for (int i = 0; i < size; i++)  // Itterate over rows
	{
		for (int j = 0; j < size; j++) // Itterate over cols
			outFile << w[i * size + j] << " ";
		if (i != size - 1)
			outFile << ";\n";
	}
	outFile.close();
}
void check(float *a, float *b)
{
	std::vector<float> vc;
	for (int i = 0; i < N*N; i++)
	{
		if (a[i] != b[i])
			vc.push_back(i);
	}
}
#ifdef SHARED_TEST

__global__ void min_kernel(float* xVal, float* xVal_window, float* out)
{
	//Declarations
	//__shared__ float xVal_Shm[256];
	__shared__ float xVal_smem[BLK_SIZE + WIN_SIZE][BLK_SIZE + WIN_SIZE]; //threadDim.x, threadDim.y size
	float mean = 0, stdev = 0, skw = 0, krt = 0, stmp = 0;
	float iB, jB;
	//https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf

	//Used in code
	int global_idx = (threadIdx.x + blockIdx.x * blockDim.x);
	int global_idy = (threadIdx.y + blockIdx.y * blockDim.y);
	if ((global_idx < N) && (global_idy < N))
	{
		xVal_smem[threadIdx.x][threadIdx.y] = *(xVal + global_idx * N + global_idy);
		if ((threadIdx.x >= (BLK_SIZE - WIN_SIZE)) && (threadIdx.y >= (BLK_SIZE - WIN_SIZE)))
		{
			xVal_smem[threadIdx.x + WIN_SIZE][threadIdx.y + WIN_SIZE] = *(xVal + (global_idx + WIN_SIZE)* N + (global_idy + WIN_SIZE));
		}
		if (threadIdx.y >= (BLK_SIZE - WIN_SIZE))
		{
			//smem[threadIdx.x][threadIdx.y + WIN_SIZE] = *((int*)kcell + idx * N + idy + WIN_SIZE);
			xVal_smem[threadIdx.x][threadIdx.y + WIN_SIZE] = *(xVal + global_idx * N + global_idy + WIN_SIZE);
			//printf("%d %d %d %d %d %d %d\n", threadIdx.x, threadIdx.y, threadIdx.x, threadIdx.y + WIN_SIZE, xVal_smem[threadIdx.x][threadIdx.y + WIN_SIZE], temp, *((int*)xVal + global_idx * N + global_idy));
		}
		if (threadIdx.x >= (BLK_SIZE - WIN_SIZE))
		{
			xVal_smem[threadIdx.x + WIN_SIZE][threadIdx.y] = *(xVal + (global_idx + WIN_SIZE) * N + global_idy);
		}
		__syncthreads();
		//printf("%d %d %d\n", threadIdx.x, threadIdx.y, temp);
	}
	if ((global_idx < N - WIN_SIZE) && (global_idy < N - WIN_SIZE))
	{
		float temp = xVal_smem[threadIdx.x][threadIdx.y];
		for (int x = 0; x < WIN_SIZE; x++)
		{
			for (int y = 0; y < WIN_SIZE; y++)
			{
				if (temp < xVal_smem[threadIdx.x + x][threadIdx.y + y])
				{
					temp = xVal_smem[threadIdx.x + x][threadIdx.y + y];
				}
			}
		}
		out[global_idx * N + global_idy] = temp;
	}
	else
	{
		out[global_idx * N + global_idy] = xVal_smem[threadIdx.x][threadIdx.y];
	}
}
#endif
void fill_input(float *a)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			a[i * N + j] = i + j;
		}
	}
}
//float init_array[(N + WIN_SIZE)][(N + WIN_SIZE)];
void min_CPU(float init_array[N*N], float target[])
{
	/*memset(init_array, 0, sizeof(init_array));
	for (int i = 0; i < N + WIN_SIZE; i++)
	{
		for (int j = 0; j < N + WIN_SIZE; j++)
		{
			if (i < N && j < N)
			{
				init_array[i][j] = src[i * N + j];
			}
			else
			{
				if (i >= N && j < N)
				{
					init_array[i][j] = init_array[(i - WIN_SIZE)][j];
				}
				else if (j >= N && i < N)
				{
					init_array[i][j] = init_array[i][(j - WIN_SIZE)];
				}
				else if (i >= N && j >= N)
				{
					init_array[i][j] = init_array[(i - WIN_SIZE)][(j - WIN_SIZE)];
				}
			}
		}
	}
	*///First copy WIN_SIZE cols and WIN_SIZE rows of the array
	for (int i = 0; i < N - WIN_SIZE; i++)
	{
		for (int j = 0; j < N - WIN_SIZE; j++)
		{
			float temp = init_array[i * N + j];
			for (int p = 0; p < WIN_SIZE; p++)
			{
				for (int q = 0; q < WIN_SIZE; q++)
				{
					if (temp < init_array[(i + p)*N +(j + q)])
						temp = init_array[(i + p) *N + (j + q)];
				}
			}
			target[i * N + j] = temp;
		}
	}
}
void kernel_wrapper()
{
#ifndef SHARED_TEST
	FILE *fp = NULL;
	fp = fopen("Xval.txt", "r");
	if (fp == NULL)
		printf("ERROR\n");
	int a[N*N]; //= (int *)malloc(sizeof(int) * N * N);
	int h_orig_out[N*N];
	int h_shared_out[N*N];
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			fscanf(fp, "%f", &a[i*N + j]);
		}
	}

	//Step 2:
	dim3 gridSize(N, N, 1);
	dim3 blockSize(N/2, N/2, 1);
	int* d_a, *d_b, *d_out;

	hipMalloc(&d_a, sizeof(int) * N*N);
	hipMalloc(&d_b, sizeof(int) * N*N);
	hipMalloc(&d_out, sizeof(int)*N*N);

	hipMemcpy(d_a, a, sizeof(int) * N*N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, a, sizeof(int) * N*N, hipMemcpyHostToDevice);
#endif
#ifdef SHARED_TEST
	FILE *fp = NULL;
	fp = fopen("Xval.txt", "r");
	if (fp == NULL)
		printf("ERROR\n");
	float a[N*N];
	fill_input(a);
	/*for (int i = 0; i < N; i++)
	{
	for (int j = 0; j < N; j++)
	{
	fscanf(fp, "%f", &a[i*N + j]);
	}
	}*/

	dim3 gridSize(4, 4, 1);
	dim3 blockSize(4, 4, 1);
	static float h_orig_out[N * N];
	float h_shared_out[N*N];
	memcpy(h_orig_out, a, sizeof(a));
	min_CPU(a, h_orig_out);
	//write_to_file_DEBUG(h_orig_out, sizeof(h_orig_out) / sizeof(float));
	float* d_a, *d_b, *d_out;
	hipMalloc(&d_a, sizeof(float) * N*N);
	hipMalloc(&d_b, sizeof(float) * N*N);
	hipMalloc(&d_out, sizeof(float)*N*N);

	hipMemset(d_a, 0, sizeof(float)* N*N);
	hipMemset(d_b, 0, sizeof(float)* N*N);
	hipMemset(d_out, 0, sizeof(float)* N*N);

	hipMemcpy(d_a, a, sizeof(float) * N*N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, a, sizeof(float) * N*N, hipMemcpyHostToDevice);
	min_kernel<< < gridSize, blockSize >> > (d_a, d_b, d_out);
	hipMemcpy(h_shared_out, d_out, sizeof(float) * N*N, hipMemcpyDeviceToHost);

#if 0
	hipMalloc(&d_a, sizeof(int) * (N + WIN_SIZE) * (N + WIN_SIZE));
	hipMalloc(&d_b, sizeof(int) * (N + WIN_SIZE) * (N + WIN_SIZE));
	hipMalloc(&d_out, sizeof(int)*N*N);

	hipMemcpy(d_a, init_array, sizeof(int) * (N + WIN_SIZE) * (N + WIN_SIZE), hipMemcpyHostToDevice);
	hipMemcpy(d_b, init_array, sizeof(int) * (N + WIN_SIZE) * (N + WIN_SIZE), hipMemcpyHostToDevice);
	min_kernel_no_shared<< < gridSize, blockSize >> > (d_a, d_b, d_out);
	hipMemcpy(h_shared_out, d_out, sizeof(int) * N*N, hipMemcpyDeviceToHost);
#endif
#endif
#if 0
	A5_fast_lo_stats_kernel << < gridSize, blockSize, 0 >> >(d_a,d_b d_out);
	hipMemcpy(h_orig_out, d_out, N*N*sizeof(int), hipMemcpyDeviceToHost);
	hipMemset(d_out, 0, N * N);
	//write_to_file_DEBUG(h_orig_out, N);
	A5_fast_lo_stats_kernel_SHARED << < gridSize, blockSize, 0 >> >(d_a, d_out);
	hipMemcpy(h_shared_out, d_out, N*N*sizeof(int), hipMemcpyDeviceToHost);
#endif
	
	check(h_orig_out, h_shared_out);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
	getchar();


	//------------------------------
	// END OF MAD ALGORITHM
	//------------------------------
	// End NVTX Marker for CUDA-MAD:
	hipDeviceReset();
	//allocate resources
	//int *d_cell; hipMalloc(&d_cell, (N*N)*sizeof(int));
	//	int *d_node; 
	//	hipMalloc(&d_node, (BLK_SIZE + WIN_SIZE) * (BLK_SIZE + WIN_SIZE)* sizeof(int));
	//	//int a[64] = {
	//	//		1, 1, 1, 1, 1, 1, 1, 1,
	//	//		1, 1, 1, 1, 1, 1, 1, 1,
	//	//		1, 1, 1, 1, 1, 1, 1, 1,
	//	//		1, 1, 1, 1, 1, 1, 1, 1,
	//	//		1, 1, 1, 1, 1, 1, 1, 1,
	//	//		1, 1, 1, 1, 1, 1, 1, 1,
	//	//		1, 1, 1, 1, 1, 1, 1, 1,
	//	//		1, 1, 1, 1, 1, 1, 1, 1};
	//		//int op[64];
	//	int a[64] = {
	//		8,7,6,5,4,3,2,1,
	//		8, 7, 6, 5, 4, 3, 2, 1,
	//		8, 7, 6, 5, 4, 3, 2, 1, 
	//		8, 7, 6, 5, 4, 3, 2, 1, 
	//		8, 7, 6, 5, 4, 3, 2, 1, 
	//		8, 7, 6, 5, 4, 3, 2, 1, 
	//		8, 7, 6, 5, 4, 3, 2, 1, 
	//		8, 7, 6, 5, 4, 3, 2, 1};
	//	int op[64];
	//	memset(op, 0, sizeof(op));
	//	dim3 blocksize(N, N, 1);
	//	hipMemcpy(d_cell, a, N * N * sizeof(int), hipMemcpyHostToDevice);
	//	//hipMemcpy(d_node, op, N * N * sizeof(int), hipMemcpyHostToDevice);
	//	compute_win2D << <1, blocksize >> >(d_node, d_cell);
	//	hipMemcpy(op, d_node, N * N* sizeof(int), hipMemcpyDeviceToHost);
	//	//free resources
	//	hipFree(d_cell); hipFree(d_node);
}
#ifdef OLD_CODE
void kernel_wrapper()
{
	//Read from xVAL
	//Use default code, calculate mean and write it to a file
	//Read from xval
	//Use the shared code, calcualte mean and write it to file.
	read_from_file();
	//allocate resources
	int *d_cell; hipMalloc(&d_cell, (N*N)*sizeof(int));
	int *d_node;
	hipMalloc(&d_node, (BLK_SIZE + WIN_SIZE) * (BLK_SIZE + WIN_SIZE)* sizeof(int));
	//int a[64] = {
	//		1, 1, 1, 1, 1, 1, 1, 1,
	//		1, 1, 1, 1, 1, 1, 1, 1,
	//		1, 1, 1, 1, 1, 1, 1, 1,
	//		1, 1, 1, 1, 1, 1, 1, 1,
	//		1, 1, 1, 1, 1, 1, 1, 1,
	//		1, 1, 1, 1, 1, 1, 1, 1,
	//		1, 1, 1, 1, 1, 1, 1, 1,
	//		1, 1, 1, 1, 1, 1, 1, 1};
	//int op[64];
	int a[64] = {
		8, 7, 6, 5, 4, 3, 2, 1,
		8, 7, 6, 5, 4, 3, 2, 1,
		8, 7, 6, 5, 4, 3, 2, 1,
		8, 7, 6, 5, 4, 3, 2, 1,
		8, 7, 6, 5, 4, 3, 2, 1,
		8, 7, 6, 5, 4, 3, 2, 1,
		8, 7, 6, 5, 4, 3, 2, 1,
		8, 7, 6, 5, 4, 3, 2, 1 };
	int op[64];
	memset(op, 0, sizeof(op));
	dim3 blocksize(N, N, 1);
	hipMemcpy(d_cell, a, N * N * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(d_node, op, N * N * sizeof(int), hipMemcpyHostToDevice);
	compute_win2D << <1, blocksize >> >(d_node, d_cell);
	hipMemcpy(op, d_node, N * N* sizeof(int), hipMemcpyDeviceToHost);
	//free resources
	hipFree(d_cell); hipFree(d_node);


	getchar();


	//------------------------------
	// END OF MAD ALGORITHM
	//------------------------------
	// End NVTX Marker for CUDA-MAD:
	hipDeviceReset();
}
#endif
//-----------------------------------------------------------------------------
